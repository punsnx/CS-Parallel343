
#include <hip/hip_runtime.h>

#define NUM_BINS 128


__global__ void histogram_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elemets, unsigned int numbins)
{
	// Privatized bins
	// To implement

	// Histogram
	// To implement

	// Commit global memory
	// To implement
}

void histogram(unsigned int* input, unsigned int* bins, 
		unsigned int num_elements, unsigned int num_bins)
{
	// Set each bin counter to zero
	// To implement

	histogram_kernel<<< 30, 512 >>>(input, bins, num_elements, num_bins);
}	

int main()
{
	int inputLength;
	unsigned int* hostInput;
	unsigned int* hostBins;
	
	unsigned int* deviceInput;
	unsigned int* deviceBins;	

	// Init hostInput & hostBins


	// Allocate device memory

	
	// Copy host memory to device


	// initiailize thread block and grid dimensions
	// invoke CUDA kernel
	histogram(deviceInput, deviceBins, inputLength, NUM_BINS);


	// Copy results from device to host

	// deallocate device memory


}
