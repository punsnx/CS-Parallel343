
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void kernelFunc(){
    printf("Hello block %u %thread %u\n", blockIdx.x,threadIdx.x);
}


int main(){
    //block,blockSize
    kernelFunc<<<9,4>>>();
    hipDeviceSynchronize();
    //see that print blocks and threads print indepentdent in SIMD
    //not sequential order 

    return 0;
}