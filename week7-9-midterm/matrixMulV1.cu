
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 3
#define BLOCK_SIZE 8
__global__ void matrixMulV1(int *d_a,int *d_b,int *d_c,int n){
    int r = blockDim.y * blockIdx.y + threadIdx.y;
    int c = blockDim.x * blockIdx.x + threadIdx.x;

    if(r < n && c < n){
        int t = 0;
        for(int i = 0;i < n;++i){
            t += d_a[r * n + i] * d_b[i * n + c];
        }
        d_c[r * n + c] = t;

    }



}
int main(){

    int n = N,blockSize = BLOCK_SIZE;
    int block = (n + blockSize - 1)/blockSize;
    dim3 dimGrid(block,block,1);
    dim3 dimBlock(blockSize,blockSize,1);
    int *h_a,*h_b,*h_c;
    h_a = (int*)malloc(n*n*sizeof(int));
    h_b = (int*)malloc(n*n*sizeof(int));
    h_c = (int*)malloc(n*n*sizeof(int));
    for(int i = 0;i<n;++i)
        for(int j = 0;j<n;++j)h_a[i * n + j] = h_b[i * n + j] = i * n + j;
    int *d_a,*d_b,*d_c;
    hipMalloc((void**) &d_a,n*n*sizeof(int));
    hipMalloc((void**) &d_b,n*n*sizeof(int));
    hipMalloc((void**) &d_c,n*n*sizeof(int));
    hipMemcpy(d_a,h_a,n*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,n*n*sizeof(int),hipMemcpyHostToDevice);

    matrixMulV1<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,n);
    hipDeviceSynchronize();
    hipMemcpy(h_c,d_c,n*n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0;i<n;++i){
        for(int j = 0;j<n;++j)
            printf("%d ",h_a[i * n + j]);
        printf("\n");
    }
    printf("\n");
    for(int i = 0;i<n;++i){
        for(int j = 0;j<n;++j)
            printf("%d ",h_b[i * n + j]);
        printf("\n");
    }
    printf("\n");

    for(int i = 0;i<n;++i){
        for(int j = 0;j<n;++j)
            printf("%d ",h_c[i * n + j]);
        printf("\n");
        
    }



    return 0;
}